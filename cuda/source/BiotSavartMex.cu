#include "hip/hip_runtime.h"
#include "mex.h"
#include "matrix.h"
#include "cutil_math.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <omp.h>
//#include "cuPrintf.cu"

// Input Arguments
#define F1_in       prhs[0]
#define F2_in       prhs[1]
#define P_in        prhs[2]
#define GAMMA_in    prhs[3]
#define RC_in       prhs[4]
#define D_in        prhs[5]
#define CMOD_in     prhs[6]
#define CO_in       prhs[7]
#define TYPE_in     prhs[8]
#define GPU_in      prhs[9]
#define GPUHW_in      prhs[10]

// Output Arguments
#define UIND_out    plhs[0]
#define L_out       plhs[1]


// Precision
#define precis      double

// CUDA stuff
#define pi          3.14159265358979324
#define tpb         128

typedef struct
        {
                double x;
                double y;
                double z;
        } vector3;

// CUDA get errors
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        mexPrintf("Cuda error: %s: %s.\n", msg,
                                  hipGetErrorString( err) );
    }                        
}

__constant__ int dev_np;
__constant__ int dev_nf;
__constant__ double cutoff;
__constant__ double cored;
__constant__ int threadsperblock;

// CUDA kernel for Biot-Savart Particle Interaction (Direct Sum Version)
__device__ void biot_p2p(double2 bfx, double2 bfy, double2 bfz, double2 brg, double2 bpxy, double1 bpz, double* uindx, double* uindy, double* uindz)
{

    double r1, r2, len, ldx, ldy, ldz, pxx1, pxx2, pyy1, pyy2, pzz1, pzz2, r1dr2,ubar, r1tr2, den;
    
    ldx=bfx.x-bfx.y;
    ldy=bfy.x-bfy.y;
    ldz=bfz.x-bfz.y;

    pxx1=bpxy.x-bfx.x;
    pxx2=bpxy.x-bfx.y;
    pyy1=bpxy.y-bfy.x;
    pyy2=bpxy.y-bfy.y;
    pzz1=bpz.x-bfz.x;
    pzz2=bpz.x-bfz.y;

    r1 = sqrt(pxx1*pxx1+pyy1*pyy1+pzz1*pzz1);
    r2 = sqrt(pxx2*pxx2+pyy2*pyy2+pzz2*pzz2);

    r1dr2=pxx1*pxx2+pyy1*pyy2+pzz1*pzz2;
    r1tr2=r1*r2;

    len=ldx*ldx + ldy*ldy + ldz*ldz; //L^2
    den=r1tr2*(r1tr2 + r1dr2) + cored*len; //cored already squared
    ubar=(brg.x*(r1+r2))*(1.000/(4.000*pi));
    ubar = ubar/den;
 
    uindx[0] += ubar*(pyy1*pzz2-pzz1*pyy2);
    uindy[0] += ubar*(pzz1*pxx2-pxx1*pzz2);
    uindz[0] += ubar*(pxx1*pyy2-pyy1*pxx2);
}


// CUDA kernel for Biot-Savart Particle Interaction (Shared Version)
__device__ vector3 biot_p2p_sh(double2 bfx, double2 bfy, double2 bfz, double2 brg, double2 bpxy, double1 bpz, vector3 uind)
{

    double r1, r2, len, ldx, ldy, ldz, pxx1, pxx2, pyy1, pyy2, pzz1, pzz2, r1dr2,ubar, r1tr2, den;
    
    ldx=bfx.x-bfx.y;
    ldy=bfy.x-bfy.y;
    ldz=bfz.x-bfz.y;

    pxx1=bpxy.x-bfx.x;
    pxx2=bpxy.x-bfx.y;
    pyy1=bpxy.y-bfy.x;
    pyy2=bpxy.y-bfy.y;
    pzz1=bpz.x-bfz.x;
    pzz2=bpz.x-bfz.y;

    r1 = sqrt(pxx1*pxx1+pyy1*pyy1+pzz1*pzz1);
    r2 = sqrt(pxx2*pxx2+pyy2*pyy2+pzz2*pzz2);

    r1dr2=pxx1*pxx2+pyy1*pyy2+pzz1*pzz2;
    r1tr2=r1*r2;

    len=ldx*ldx + ldy*ldy + ldz*ldz; //L^2
    den=r1tr2*(r1tr2 + r1dr2) + cored*len; //cored already squared
    ubar=(brg.x*(r1+r2))*(1.000/(4.000*pi));
    ubar = ubar/den;
 
    uind.x += ubar*(pyy1*pzz2-pzz1*pyy2);
    uind.y += ubar*(pzz1*pxx2-pxx1*pzz2);
    uind.z += ubar*(pxx1*pyy2-pyy1*pxx2);

    return uind;
}

__global__ void BiotSavart_naive(double2 *p, double1 *pz, double2 *fx, double2 *fy, double2 *fz, double2 *rg, double2 *uind, double1 *uindz)
{
//Get thread's global index
int k;
int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if(idx<dev_np){
    //Loop over all source filaments
    for(k=0; k < dev_nf; k++)
    {
      biot_p2p(fx[k], fy[k], fz[k], rg[k], p[idx], pz[idx], &uind[idx].x, &uind[idx].y, &uindz[idx].x);
    }
  }

}

__global__ void BiotSavart_sh(double2 *p, double1 *pz, double2 *fx, double2 *fy, double2 *fz, double2 *rg, double2 *uind, double1 *uindz)
{
  __shared__ double2 shFx[tpb];
  __shared__ double2 shFy[tpb];
  __shared__ double2 shFz[tpb];
  __shared__ double2 shRg[tpb];

//Get thread's global index
int k,i,j,tile;
int gid = blockIdx.x * blockDim.x + threadIdx.x;
int tilesize = blockDim.x;
int tileloc = threadIdx.x;

//Local Vars
vector3 myUind;
int fid, tileid;


    //Loop over all tiles of source filaments (tile size = block size)
    for(k=0, tile=0; k < dev_nf; k+= tilesize, tile++)
    {
      tileid = tile*tilesize+tileloc;
      fid = tile*tilesize+k;
         
      shFx[threadIdx.x] = fx[tileid];
      shFy[threadIdx.x] = fy[tileid];
      shFz[threadIdx.x] = fz[tileid];
      shRg[threadIdx.x] = rg[tileid];
     

      // Syncronize threads before using shared mem
      __syncthreads();
 
      if(gid<dev_np){
        for(i=0; i<tilesize; i++)
        {
        fid = tile*tilesize+i;
        if(fid<dev_nf)
        myUind = biot_p2p_sh(shFx[i], shFy[i], shFz[i], shRg[i], p[gid], pz[gid], myUind);
        }
      }
      __syncthreads(); 
    }
  uind[gid].x = myUind.x;
  uind[gid].y = myUind.y;
  uindz[gid].x = myUind.z;
}


// CUDA kernel that calculates segment length
__global__ void calcLengthOnly(double2 *a, double2 *b, double2 *c, double1 *d)
{
    //Get thread's global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx<dev_np){
       d[idx].x = sqrt((a[idx].x-a[idx].y)*(a[idx].x-a[idx].y) + (b[idx].x-b[idx].y)*(b[idx].x-b[idx].y) + (c[idx].x-c[idx].y)*(c[idx].x-c[idx].y));      
    }

}


// CUDA Device properties and problem size calc
static int cuda_setup(int probsize, int numthreads, int numcards, int devicenum)
{
    hipDeviceProp_t prop;
    double maxglobal;
    double maxshared;
    int blocksize;

    hipGetDeviceProperties(&prop, devicenum);
    //mexPrintf("Device Name: %s \n",prop.name);
    //mexPrintf("Global Memory: %d \n",prop.totalGlobalMem);
    //mexPrintf("Shared Memory Per Block: %d \n",prop.sharedMemPerBlock);
    //mexPrintf("Number of MPs: %d \n",prop.multiProcessorCount);   
 
    maxglobal = prop.totalGlobalMem/sizeof(double);
    //mexPrintf("Max number of elements in Global: %3.9f \n",maxglobal);
    maxshared = prop.sharedMemPerBlock/sizeof(double);
    //mexPrintf("Max number of elements in Shared Per Block: %3.9f \n",maxshared);

    blocksize = (probsize+(numthreads-1))/numthreads;
    //mexPrintf("Given thread count: %d \n",numthreads);
    //mexPrintf("Computed Block Size: %d \n",blocksize);  
    return blocksize;
}



void biot_p2p_nogpu(double2 bfx, double2 bfy, double2 bfz, double2 brg, double2 bpxy, double1 bpz, double* uindx, double* uindy, double* uindz, double cd, double co)
{
    //mexPrintf("Cored: %3.9f \n",cd);
    double r1, r2, len, ldx, ldy, ldz, pxx1, pxx2, pyy1, pyy2, pzz1, pzz2, r1dr2,ubar, r1tr2, den;
    
    ldx=bfx.x-bfx.y;
    ldy=bfy.x-bfy.y;
    ldz=bfz.x-bfz.y;

    pxx1=bpxy.x-bfx.x;
    pxx2=bpxy.x-bfx.y;
    pyy1=bpxy.y-bfy.x;
    pyy2=bpxy.y-bfy.y;
    pzz1=bpz.x-bfz.x;
    pzz2=bpz.x-bfz.y;

    r1 = sqrt(pxx1*pxx1+pyy1*pyy1+pzz1*pzz1);
    r2 = sqrt(pxx2*pxx2+pyy2*pyy2+pzz2*pzz2);

    r1dr2=pxx1*pxx2+pyy1*pyy2+pzz1*pzz2;
    r1tr2=r1*r2;

    len=ldx*ldx + ldy*ldy + ldz*ldz; //L^2
    den=r1tr2*(r1tr2 + r1dr2) + cd*len; //cored already squared
    ubar=(brg.x*(r1+r2))*(1.000/(4.000*pi));
    ubar = ubar/den;
 
    uindx[0] += ubar*(pyy1*pzz2-pzz1*pyy2);
    uindy[0] += ubar*(pzz1*pxx2-pxx1*pzz2);
    uindz[0] += ubar*(pxx1*pyy2-pyy1*pxx2);
}

void BiotSavart_nogpu(double2 *p, double1 *pz, double2 *fx, double2 *fy, double2 *fz, double2 *rg, double2 *uind, double1 *uindz, int n, int f_n, double cd, double co)
{
omp_set_num_threads(4);
int j,k,omptnum;
  // Loop over all particles  
  for(j=0; j < n; j++)
  {
    //Loop over all source filaments
    for(k=0; k < f_n; k++)
    {
      biot_p2p_nogpu(fx[k], fy[k], fz[k], rg[k], p[j], pz[j], &uind[j].x, &uind[j].y, &uindz[j].x, cd, co);
    }
  }
  
}



// Resolve arrays to working 3D matrix
static void crunch_array(double *arrayin, double3 *arrayout, mwSize *di)
{
   
    mwSize ns = di[0];
    mwSize nd = di[1];
    mwSize nt = di[2];
    mwSize nb = di[3];

    int i = 0;
    int m,n,p;
    
    

        for(p=0; p<nb;p++){ 
          for(n=0; n<nt; n++){ 
             for(m=0; m<ns; m++){
                 
            arrayout[i].x = arrayin[m+0*ns+n*ns*nd+p*ns*nt*nd];
            arrayout[i].y = arrayin[m+1*ns+n*ns*nd+p*ns*nt*nd];
            arrayout[i].z = arrayin[m+2*ns+n*ns*nd+p*ns*nt*nd];
          
              i++;
            }
        }
    }
}


// Two input arrays gamma and rc stuffed into double2
static void crunch_rg(double *arrayin1, double *arrayin2, double2 *arrayout, mwSize *di)
{
    mwSize ns = di[0];
    mwSize nd = 1;
    mwSize nt = di[2];
    mwSize nb = di[3];
    int i = 0;
    int m,n,p;
        for(p=0; p<nb;p++){ 
          for(n=0; n<nt; n++){ 
            for(m=0; m<ns; m++){     
            arrayout[i].x = arrayin1[m+0*ns+n*ns*nd+p*ns*nt*nd];
            arrayout[i].y = arrayin2[m+0*ns+n*ns*nd+p*ns*nt*nd];
            i++;
            }
        }
    }
}

// Two input arrays F1 and F2 stuffed into 3x double2
static void crunch_f(double *arrayin1, double *arrayin2, double2 *arrayout1,double2 *arrayout2,double2 *arrayout3, mwSize *di)
{
    mwSize ns = di[0];
    mwSize nd = di[1];
    mwSize nt = di[2];
    mwSize nb = di[3];
    int i = 0;
    int m,n,p;
        for(p=0; p<nb;p++){ 
          for(n=0; n<nt; n++){ 
            for(m=0; m<ns; m++){     
            arrayout1[i].x = arrayin1[m+0*ns+n*ns*nd+p*ns*nt*nd];
            arrayout1[i].y = arrayin2[m+0*ns+n*ns*nd+p*ns*nt*nd];
            arrayout2[i].x = arrayin1[m+1*ns+n*ns*nd+p*ns*nt*nd];
            arrayout2[i].y = arrayin2[m+1*ns+n*ns*nd+p*ns*nt*nd];
            arrayout3[i].x = arrayin1[m+2*ns+n*ns*nd+p*ns*nt*nd];
            arrayout3[i].y = arrayin2[m+2*ns+n*ns*nd+p*ns*nt*nd];
            i++;
            }
        }
    }
}


// One input array P stuffed into 1x double2 1x double1
static void crunch_p(double *arrayin1, double2 *arrayout1,double1 *arrayout2,mwSize *di)
{
    mwSize ns = di[0];
    mwSize nd = 3;
    mwSize nt = di[2];
    mwSize nb = di[3];
    int i = 0;
    int m,n,p;
        for(p=0; p<nb;p++){ 
          for(n=0; n<nt; n++){ 
            for(m=0; m<ns; m++){     
            arrayout1[i].x = arrayin1[m+0*ns+n*ns*nd+p*ns*nt*nd];
            arrayout1[i].y = arrayin1[m+1*ns+n*ns*nd+p*ns*nt*nd];
            arrayout2[i].x = arrayin1[m+2*ns+n*ns*nd+p*ns*nt*nd];
            i++;
            }
        }
    }
}


// Reassemble uind
static void reassem_uind(double *arrayout, double2 *arrayin1,double1 *arrayin2,mwSize *di)
{
    mwSize ns = di[0];
    mwSize nd = 3;
    mwSize nt = di[2];
    mwSize nb = di[3];
    int i = 0;
    int m,n,p;
        for(p=0; p<nb;p++){ 
          for(n=0; n<nt; n++){ 
            for(m=0; m<ns; m++){     
            arrayout[m+0*ns+n*ns*nd+p*ns*nt*nd] = arrayin1[i].x;
            arrayout[m+1*ns+n*ns*nd+p*ns*nt*nd] = arrayin1[i].y;
            arrayout[m+2*ns+n*ns*nd+p*ns*nt*nd] = arrayin2[i].x;
            i++;
            }
        }
    }
}

// Reassemble double3 into 4d array
static void reassem_array3(double3 arrayin[], double arrayout[], mwSize *di)
{
    mwSize ns = di[0];
    mwSize nd = di[1];
    mwSize nt = di[2];
    mwSize nb = di[3];

    int i = 0;
    int m,n,p;
    
    

        for(p=0; p<nb;p++){ 
          for(n=0; n<nt; n++){ 
             for(m=0; m<ns; m++){
                 
            arrayout[m+0*ns+n*ns*nd+p*ns*nt*nd] = arrayin[i].x;

              i++;
            }
        }
    }
}

// Reassemble double3 into 4d array
static void reassem_array1(double1 arrayin[], double arrayout[], mwSize *di)
{
    mwSize ns = di[0];
    mwSize nd = 1;
    mwSize nt = di[2];
    mwSize nb = di[3];

    int i = 0;
    int m,n,p;

        for(p=0; p<nb;p++){ 
          for(n=0; n<nt; n++){ 
             for(m=0; m<ns; m++){
                 
            arrayout[m+0*ns+n*ns*nd+p*ns*nt*nd] = arrayin[i].x;

              i++;
            }
        }
    }
}

// Biot Savart Launching Routine
static void biot_calc(double *uind ,double *L, double *F1, double *F2,
                   double *P, double *gamma, double *rc, double d, char *cm, 
                   double co, char *casetype, mwSize ndi, mwSize *di, 
                   mwSize *fdi, char *usegpu, double *gpuinfo)
    {
      
    // Dimensions
    mwSize ns = di[0];
    mwSize nd = di[1];
    mwSize nt = di[2];
    mwSize nb = di[3];

    mwSize f_ns = fdi[0];
    mwSize f_nd = fdi[1];
    mwSize f_nt = fdi[2];
    mwSize f_nb = fdi[3];

    // Various problem constants and indices
    mwSize cartprobsize = ns*nt*nb;
    mwSize f_cartprobsize = f_ns*f_nt*f_nb;
    int np = (int)cartprobsize;
    int nf = (int)f_cartprobsize;
    int i,j,k;

    // Create vars to send to GPU (double2 is max native)
    double2 *pxy;
    double1 *pz;
    double2 *fx, *fy, *fz ;
    double2 *rg;
    double2 *uxy;
    double1 *uz;
    double1 *L_new;
    
    // Test whether this is a GPU enabled simulation  
    int gpusim = 0;
    if(strncmp("true",usegpu,4)==0){
       gpusim = 1;
    }

    // GPU Info
    int threadcount = (int)gpuinfo[0];
    int blockcount = 0;
    int numcards = (int)gpuinfo[1]; // Number of GPUs
    int deviceind = 0; // Only one GPU now, change for multiple
    size_t memfree, memtotal;
    clock_t t0,t1;
     
    // Create vars that will be allocated on GPU
    double2 *dev_pxy;
    double1 *dev_pz;
    double2 *dev_fx,*dev_fy, *dev_fz;
    double2 *dev_rg;
    double1 *dev_len;
    double2 *dev_uxy;
    double1 *dev_uz;
    

    //GPU Error catcher
    hipError_t hipError_t;

    // Allocate memory for local host variables
    pxy = (double2 *)malloc(cartprobsize*sizeof(double2));
    pz = (double1 *)malloc(cartprobsize*sizeof(double1));
    uxy = (double2 *)malloc(cartprobsize*sizeof(double2));
    uz = (double1 *)malloc(cartprobsize*sizeof(double1));
    fx = (double2 *)malloc(f_cartprobsize*sizeof(double2));
    fy = (double2 *)malloc(f_cartprobsize*sizeof(double2));
    fz = (double2 *)malloc(f_cartprobsize*sizeof(double2));
    rg = (double2 *)malloc(f_cartprobsize*sizeof(double2));
    L_new = (double1 *)malloc(cartprobsize*sizeof(double1));
   
    // Pack arrays into double2 structures
    crunch_p(P,pxy,pz,di);
    crunch_f(F1,F2,fx,fy,fz,fdi);
    crunch_rg(gamma,rc,rg,fdi);

        
     
    if(gpusim == 1){
        // Check for any CUDA errors
        checkCUDAError("memcpy");
    
        // Calculate number of blocks
        blockcount = cuda_setup(f_cartprobsize, threadcount, numcards, deviceind);
      
        // Allocate memory on GPU
        hipError_t = hipMalloc( (void**)&dev_fx, f_cartprobsize*sizeof(double2) );
        if (hipError_t != hipSuccess) { mexErrMsgTxt("Out of Nvidia device memory."); }
        hipError_t = hipMalloc( (void**)&dev_fy, f_cartprobsize*sizeof(double2) );
        if (hipError_t != hipSuccess) { mexErrMsgTxt("Out of Nvidia device memory."); }
        hipError_t = hipMalloc( (void**)&dev_fz, f_cartprobsize*sizeof(double2) );
        if (hipError_t != hipSuccess) { mexErrMsgTxt("Out of Nvidia device memory."); }
        hipError_t = hipMalloc( (void**)&dev_len, cartprobsize*sizeof(double1) );
        if (hipError_t != hipSuccess) { mexErrMsgTxt("Out of Nvidia device memory."); }
        hipError_t = hipMalloc( (void**)&dev_pxy, cartprobsize*sizeof(double2) );
        if (hipError_t != hipSuccess) { mexErrMsgTxt("Out of Nvidia device memory."); }
        hipError_t = hipMalloc( (void**)&dev_pz, cartprobsize*sizeof(double1) );
        if (hipError_t != hipSuccess) { mexErrMsgTxt("Out of Nvidia device memory."); }
        hipError_t = hipMalloc( (void**)&dev_rg, f_cartprobsize*sizeof(double2) );
        if (hipError_t != hipSuccess) { mexErrMsgTxt("Out of Nvidia device memory."); }
        hipError_t = hipMalloc( (void**)&dev_uxy, cartprobsize*sizeof(double2) );
        if (hipError_t != hipSuccess) { mexErrMsgTxt("Out of Nvidia device memory."); }
        hipError_t = hipMalloc( (void**)&dev_uz, cartprobsize*sizeof(double1) );
        if (hipError_t != hipSuccess) { mexErrMsgTxt("Out of Nvidia device memory."); }
        
        // Reset GPU memory blocks to 0
        hipMemset(dev_uxy, 0.00, cartprobsize*sizeof(double2));
        hipMemset(dev_uz, 0.00, cartprobsize*sizeof(double1));
        hipMemset(dev_fx, 0.00, f_cartprobsize*sizeof(double2));
        hipMemset(dev_fy, 0.00, f_cartprobsize*sizeof(double2));
        hipMemset(dev_fz, 0.00, f_cartprobsize*sizeof(double2));

        // Copy host memory vars to device memory vars
        hipMemcpy(dev_fx, fx, f_cartprobsize*sizeof(double2), hipMemcpyHostToDevice);
        hipMemcpy(dev_fy, fy, f_cartprobsize*sizeof(double2), hipMemcpyHostToDevice);  
        hipMemcpy(dev_fz, fz, f_cartprobsize*sizeof(double2), hipMemcpyHostToDevice); 
        hipMemcpy(dev_pxy, pxy, cartprobsize*sizeof(double2), hipMemcpyHostToDevice); 
        hipMemcpy(dev_pz, pz, cartprobsize*sizeof(double1), hipMemcpyHostToDevice); 
        hipMemcpy(dev_rg, rg, f_cartprobsize*sizeof(double2), hipMemcpyHostToDevice); 
        hipMemcpyToSymbol(HIP_SYMBOL(cutoff), &co, sizeof(double));
        hipMemcpyToSymbol(HIP_SYMBOL(cored), &d, sizeof(double));
        hipMemcpyToSymbol(HIP_SYMBOL(dev_np), &np, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(dev_nf), &nf, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(threadsperblock), &blockcount, sizeof(int));


        // Check for any CUDA errors
        checkCUDAError("memcpy");
       
        // Length calculation (Testing only...not used)
        if(strncmp("leng",casetype,4)==0){
        //mexPrintf("Calling the length cuda kernel!\n");
        dim3 dimGrid(blockcount);
        dim3 dimBlock(threadcount);
            calcLengthOnly<<<dimGrid,dimBlock>>>(dev_fx, dev_fy, dev_fz, dev_len);       
        }
        
        // Biot-Savart naive kernel call
        double c0 = 0; 
        if(strncmp("fuln",casetype,4)==0){
        //mexPrintf("Calling the full cuda kernel!\n");
        dim3 dimGrid(blockcount);
        dim3 dimBlock(threadcount);
        c0 = omp_get_wtime( );
            BiotSavart_naive<<<dimGrid,dimBlock>>>(dev_pxy, dev_pz, dev_fx, dev_fy, dev_fz, dev_rg, dev_uxy, dev_uz);       
         
        }
        
        // Biot-Savart shared memory kernel
        if(strncmp("fuls",casetype,4)==0){
        //mexPrintf("Calling the full cuda kernel!\n");
        dim3 dimGrid(blockcount);
        dim3 dimBlock(threadcount);
        c0 = omp_get_wtime( );
            BiotSavart_sh<<<dimGrid,dimBlock>>>(dev_pxy, dev_pz, dev_fx, dev_fy, dev_fz, dev_rg, dev_uxy, dev_uz);       
         
        }
               
        // block until the device has completed
        hipDeviceSynchronize();
        double c1 = omp_get_wtime( );
        //mexPrintf ("Elapsed wall clock time: %3.9f seconds\n", c1-c0); 

        // Check for any CUDA errors
        checkCUDAError("kernel invocation");
 
        if(strncmp("leng",casetype,4)==0){
        hipMemcpy(L_new, dev_len, cartprobsize*sizeof(double1), hipMemcpyDeviceToHost); 
        }

        if(strncmp("ful",casetype,3)==0){
        hipMemcpy(uxy, dev_uxy, cartprobsize*sizeof(double2), hipMemcpyDeviceToHost);
        hipMemcpy(uz, dev_uz, cartprobsize*sizeof(double1), hipMemcpyDeviceToHost);  
         }


        // Check for any CUDA errors
        checkCUDAError("memcpy");


    } //Close if GPU Sim

    if(gpusim == 0){
         
         
         memset(uxy, 0.00, cartprobsize*sizeof(double2));
         memset(uz, 0.00, cartprobsize*sizeof(double1));

         BiotSavart_nogpu(pxy, pz, fx, fy, fz, rg, uxy, uz, cartprobsize, f_cartprobsize, d, co);
         
         
    }
    
    if(strncmp("leng",casetype,4)==0){
    reassem_array1(L_new,L,di);
    }
    
    if(strncmp("ful",casetype,3)==0){
    reassem_uind(uind,uxy,uz,di);  
    }
    
    
 
    if(gpusim == 1){
    hipFree(dev_uxy);
    hipFree(dev_uz);
    hipFree(dev_fx);
    hipFree(dev_fy);
    hipFree(dev_fz);
    hipFree(dev_len);
    hipFree(dev_pxy);
    hipFree(dev_pz);
    hipFree(dev_rg);
    }
    
    free(pxy);
    free(pz);
    free(uxy);
    free(uz);
    free(fx);
    free(fy);
    free(fz);
    free(L_new);
    free(rg);
    
  
    if(gpusim == 1){
    hipMemGetInfo(&memfree, &memtotal);
    //mexPrintf("Free Memory: %d \n",memfree);
    //mexPrintf("Total Memory: %d \n",memtotal);
    }

    return;
    }

// Function which interfaces with MATLAB
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{
    
    mwSize ndims, ndimsin;
    mwSize dims[4],lendims[4], fdims[4];
    const mwSize *indims, *findims;
    double *uind_ptr, *l_ptr;
    double *f1_ptr, *f2_ptr, *p_ptr, *gamma_ptr;
    double *rc_ptr, *co_ptr, *d_ptr;
    double *gpuhw_ptr;
    char *type_ptr, *cm_ptr, *gpuflag_ptr;
    const char *cmnum_ptr;
    mwSize cm_buflen, type_buflen, gpuflag_buflen, cmnum_buflen;

    cm_buflen = mxGetNumberOfElements(CMOD_in) + 1; 
    type_buflen = mxGetNumberOfElements(TYPE_in) + 1; 
    gpuflag_buflen = mxGetNumberOfElements(GPU_in) + 1; 
    cmnum_buflen = cm_buflen - 4; 
 
    cm_ptr = (char *)mxCalloc(cm_buflen, sizeof(char));
    type_ptr = (char *)mxCalloc(type_buflen, sizeof(char));
    gpuflag_ptr = (char *)mxCalloc(gpuflag_buflen, sizeof(char));
    cmnum_ptr = (const char *)mxCalloc(cmnum_buflen, sizeof(char));
    

    if (mxGetString(CMOD_in, cm_ptr, cm_buflen))
        mexErrMsgTxt("Couldn't make Core Model string.");
    if (mxGetString(TYPE_in, type_ptr, type_buflen))
        mexErrMsgTxt("Couldn't make TYPE string.");
    if (mxGetString(GPU_in, gpuflag_ptr, gpuflag_buflen))
        mexErrMsgTxt("Couldn't make GPU string.");

    if (nrhs != 11)
        mexErrMsgTxt("11 inputs expected.");
    if (nlhs != 2)
        mexErrMsgTxt("2 outputs expected."); 
    
    //Copy Core model Number to Int
    
    

    ndimsin = mxGetNumberOfDimensions(P_in);
    //mexPrintf("Num Dims In: %i \n",ndimsin);


    //mexPrintf("1st Type is: %s\n", type_ptr);
    
    if(ndimsin != 4){
      //mexPrintf("Changing Number of Dimensions to 4!\n");
      ndims = 4;
    }else{
    ndims = ndimsin;
    }
    
    indims = mxGetDimensions(P_in);
    findims = mxGetDimensions(F1_in);    

    if(ndimsin == 1){
         mexErrMsgTxt("Need more dimensions in P.");
    }else if(ndimsin == 2){
         dims[0] = indims[0];
         dims[1] = indims[1];
         dims[2] = 1;
         dims[3] = 1;
    }else if(ndimsin == 3){
         dims[0] = indims[0];
         dims[1] = indims[1];
         dims[2] = indims[2];
         dims[3] = 1;
    }else if(ndimsin == 4){
         dims[0] = indims[0];
         dims[1] = indims[1];
         dims[2] = indims[2];
         dims[3] = indims[3]; 
    }else{
         mexErrMsgTxt("Wrong dimensions in P.");
    } 

    fdims[0] = findims[0];
    fdims[1] = findims[1];
    fdims[2] = findims[2];
    fdims[3] = findims[3]; 

    //mexPrintf("Dims: %i %i %i %i \n",dims[0],dims[1],dims[2],dims[3]);
    //mexPrintf("F Dims: %i %i %i %i \n",fdims[0],fdims[1],fdims[2],fdims[3]);
    //mexPrintf("Num Dims: %i \n",ndims);

    // Create a matrix for the return arguments
    UIND_out = mxCreateNumericArray(ndims, dims, mxDOUBLE_CLASS, mxREAL);
    
         lendims[0] = dims[0];
         lendims[1] = 1;
         lendims[2] = dims[2];
         lendims[3] = dims[3]; 
  
    L_out = mxCreateNumericArray(ndims, lendims, mxDOUBLE_CLASS, mxREAL);
   
    // Assign pointers to the various parameters
    uind_ptr = mxGetPr(UIND_out);
    l_ptr = mxGetPr(L_out);     
    f1_ptr = mxGetPr(F1_in); 
    f2_ptr = mxGetPr(F2_in);
    p_ptr = mxGetPr(P_in);
    gamma_ptr = mxGetPr(GAMMA_in);
    rc_ptr = mxGetPr(RC_in);
    d_ptr = mxGetPr(D_in);
    co_ptr = mxGetPr(CO_in);
    gpuhw_ptr = mxGetPr(GPUHW_in);
    

 
        
    // Do the actual computations in the biot subroutine
    biot_calc(uind_ptr,l_ptr,f1_ptr,f2_ptr,p_ptr,
              gamma_ptr,rc_ptr,d_ptr[0],cm_ptr,co_ptr[0],
              type_ptr, ndims, dims, fdims, gpuflag_ptr, gpuhw_ptr); 
    

    
}

    